#include "hip/hip_runtime.h"

#include "cellSchAlgGPU.cuh"
#include <vector>
#include <cmath>
#include <fstream>
#include <iostream>
#include <chrono>
#include <array>

// CUDA相关头文件
#include <hip/hip_runtime.h>
#include <>

// #include <pybind11/pybind11.h>
// #include <pybind11/stl.h>

namespace dpnblist
{

    __global__ void initializeArray(int *array, int size) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < size) {
            array[tid] = -1;
        }
    }
    /*
    * to get the neighbor cell index of each cell, and store them in nebcell_list
    * param: offset_vec: the offset vector of each neighbor cell
    * param: ncells: the number of cells
    * param: n_nebcells: the number of neighbor cells
    * param: L: the box length of each dimension
    */
    __global__ void get_neb(int* offset_vec, int *ncells, int *n_nebcells, int *L, int* nebcell) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        if (tid < *ncells) {
            // int L[3] = {4, 4, 4};
            int cell_veci[3];
            int offset_veci[3];
            float neb_vec[3];
            float wrap_neb_vec[3];
            int round_wrap_neb_vec[3];
            float f[3];
            float wrapped_f[3];

            cell_veci[0] = tid / (L[1] * L[2]);
            cell_veci[1] = (tid - cell_veci[0] * L[1] * L[2]) / L[2];
            cell_veci[2] = tid - cell_veci[0] * L[1] * L[2] - cell_veci[1] * L[2];

            for (int i = 0; i < *n_nebcells; ++i) {
                for (int j = 0; j < 3; ++j) {
                    offset_veci[j] = offset_vec[i * 3 + j];
                    neb_vec[j] = cell_veci[j] + offset_veci[j];
                }
                for (int k = 0; k < 3; ++k) {
                    f[k] = neb_vec[k] / L[k];
                    wrapped_f[k] = f[k] - std::floor(f[k] + 0.000001);
                    wrap_neb_vec[k] = wrapped_f[k] * L[k];
                    round_wrap_neb_vec[k] = round(wrap_neb_vec[k]);
                }
                
                nebcell[*n_nebcells * tid + i] = round_wrap_neb_vec[0] * L[1] * L[2] + round_wrap_neb_vec[1] * L[2] + round_wrap_neb_vec[2];
            }
        }
    }
    /*
    * to build the linked list, and store the head, lscl, atom_cellindex, cell_count
    * param: xyz_dev: the coordinates of all atoms
    * param: head_dev: the head atom of each cell
    * param: lscl_dev: the next atom of each cell
    * param: atom_cellindex: the cell index of each atom
    * param: cell_count_dev: the number of atoms in each cell
    * param: r_cutoff: the cutoff radius
    * param: L: the box length of each dimension
    * param: natoms: the number of atoms
    */
    __global__ void build_linked_list_kernel(float *xyz_dev, int *head_dev, int *lscl_dev, int *atom_cellindex, int *cell_count_dev, float *_r_cutoff, int *L, int *natoms) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < *natoms) {
            int cell_index_vec[3];
            for (int j = 0; j < 3; j++) {
                cell_index_vec[j] = xyz_dev[i * 3 + j] / *_r_cutoff;
                if (cell_index_vec[j] == L[j])
                    cell_index_vec[j] = cell_index_vec[j] - 1;
            }
            int cell_index = cell_index_vec[0] * L[1] * L[2] + cell_index_vec[1] * L[2] + cell_index_vec[2];
            atom_cellindex[i] = cell_index;
            lscl_dev[i] = atomicExch(&head_dev[cell_index], i);
            atomicAdd(&cell_count_dev[cell_index], 1);
        }
    }

    // calculate the distance between every two element in pre_xyz and vec3_xyz, and to judge whether to update the neighbor list array
    __global__ void calc_vec_distance(float *pre_xyz, float *vec3_xyz, float *dis, int *natoms) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < *natoms) {
            float dr[3] = {vec3_xyz[i*3] - pre_xyz[i*3], vec3_xyz[i*3+1] - pre_xyz[i*3+1], vec3_xyz[i*3+2] - pre_xyz[i*3+2]};
            dis[i] = dr[0]*dr[0] + dr[1]*dr[1] + dr[2]*dr[2];
        }
    }

    // wrap the coordinates of all atoms
    __global__ void wrap_coord(float *xyz, float *wrap_xyz, float *box_len, int *natoms) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < *natoms) {
            float f[3], wrapped_f[3];
            for (int j = 0; j < 3; j++) {
                f[j] = xyz[i * 3 + j] / box_len[j];
                wrapped_f[j] = f[j] - std::floor(f[j] + 0.000001);
                wrap_xyz[i * 3 + j] = wrapped_f[j] * box_len[j];
            }
        }
    }

    /*
    * to calculate the distance between two atoms, and return the square of the distance
    * param: pos_i: the coordinates of atom i
    * param: pos_j: the coordinates of atom j
    * param: length: the box length of each dimension
    */
    __device__ float calc_distance(float *pos_i, float *pos_j, float *length) {
        float difference[3] = {0.0, 0.0, 0.0};
        float diff = 0.0;

        for (int i = 0; i < 3; i++) {
            float dri = pos_i[i] - pos_j[i];
            diff = fmodf((dri + length[i] / 2), length[i]);
            if (diff < 0) diff += length[i];
            diff -= length[i] / 2;
            difference[i] = diff;
        }

        return (difference[0]*difference[0] + difference[1]*difference[1] + difference[2]*difference[2]);
    }

    /*
    * to build the neighbor list array, and store the neighbor list of each atom to neighborListArray
    * param: natoms: the number of atoms
    * param: xyz: the coordinates of all atoms
    * param: head: the head atom of each cell
    * param: lscl: the next atom of each cell
    * param: atom_cellindex: the cell index of each atom
    * param: cell_count: the number of atoms in each cell
    * param: nebcell: the neighbor cell index of each cell
    * param: L: the box length of each dimension
    * param: box_length: the cell number of each dimension
    * param: r_cutoff2: the square of cutoff radius
    * param: neighborListArray: the neighbor list of each atom
    */
    __global__ void  buildListArray(int *natoms, float *xyz, int *head, int *lscl, int *atom_cellindex, int *cell_atoms_count, int *nebcell, int *L, float *box_length, float *r_cutoff2, int *neighborListArray, int d_nnebs){
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < *natoms){
            int count = 0;                                                   //记录这是第几个邻居原子
            float pos_i[3] = {xyz[tid*3], xyz[tid*3+1], xyz[tid*3+2]};          //原子坐标
            int cell_index = atom_cellindex[tid];                            //原子属于哪个单元
            int neighbor_cell = 0;                                           //单元的邻居单元索引
            for (int i = cell_index * 27; i < cell_index * 27 + 27; ++i)        // Loop through the neighboring cells
            {                                                                   
                neighbor_cell = nebcell[i];                                     //27个邻居单元索引
                int num_atoms = cell_atoms_count[neighbor_cell];             //单元有几个原子
                int atom_id = head[neighbor_cell];                           //单元的第一个原子
                for (int j = 0; j < num_atoms; ++j)                          //循环单元内所有的原子
                {
                    float pos_j[3] = {xyz[atom_id*3], xyz[atom_id*3+1], xyz[atom_id*3+2]};
                    float r = calc_distance(pos_i, pos_j, box_length);
                    if (r - *r_cutoff2 < 1e-7)
                    {
                        neighborListArray[tid * d_nnebs + count] = atom_id;                   // Add j to the neighbor list of i
                        count++;
                    }
                    atom_id = lscl[atom_id];
                }
            }
        }
    }

    /*
    * the constructor of class CudaCellList, to initialize the parameters
    * param: box: the box of the system
    * param: r_cutoff: the cutoff radius
    * param: skin: the skin of the system
    * initialize the parameters of the system and allocate the memory of the device
    * param: r_cutoff: the cutoff radius
    * param: skin: the skin of the system
    * param: d_box_len: the box length of each dimension
    * param: d_cell_len: the cell number of each dimension
    * param: d_ncells: the number of cells
    */
    
    CellSearchGPU::CellSearchGPU(const Box& box, const std::vector<std::vector<float>>& xyz, float r_cutoff) : _box(box), _xyz(xyz), _r_cutoff(r_cutoff)
	{
        // nnebs = 500;

        std::array<float,3> box_length = box.get_lengths_cpu3();
        // float volume = box_length[0] * box_length[1] * box_length[2];
        // float pre_nnebs = xyz.size() / volume * 4 * 3.1415926 * r_cutoff * r_cutoff * r_cutoff / 3;
        // nnebs = int(pre_nnebs * 1.11 + 61);
        nnebs = static_cast<int>(std::ceil(1.5 * (5 * xyz.size() / (box_length[0] * box_length[1] * box_length[2])) * r_cutoff * r_cutoff * r_cutoff + 45));
        if (nnebs < 100) nnebs = 100;
        for (int i = 0; i < 3; ++i) {
            _box_len[i] = box_length[i];
        }

        hipMalloc((void**)&d_box_len, 3 * sizeof(float));
        hipMemcpy(d_box_len, _box_len, 3 * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc((void**)&d_r_cutoff, sizeof(float));
        hipMemcpy(d_r_cutoff, &_r_cutoff, sizeof(float), hipMemcpyHostToDevice);

        std::vector<std::vector<int>> off_set_vec = {
            {-1, -1, -1},{-1, -1, 0},{-1, -1, 1},{-1, 0, -1},{-1, 0, 0},{-1, 0, 1},{-1, 1, -1},{-1, 1, 0},{-1, 1, 1},
            {0, -1, -1},{0, -1, 0},{0, -1, 1},{0, 0, -1},{0, 0, 0},{0, 0, 1},{0, 1, -1},{0, 1, 0},{0, 1, 1},
            {1, -1, -1},{1, -1, 0},{1, -1, 1},{1, 0, -1},{1, 0, 0},{1, 0, 1},{1, 1, -1},{1, 1, 0},{1, 1, 1}
        };
        n_nebcells = off_set_vec.size();
        // int *d_n_nebcells;
        hipMalloc((void**)&d_n_nebcells, sizeof(int));
        hipMemcpy(d_n_nebcells, &n_nebcells, sizeof(int), hipMemcpyHostToDevice);

        int off_set_vec_1d[n_nebcells * 3];
        for (int i = 0; i < n_nebcells; ++i) {
            for (int j = 0; j< 3; ++j) {
                off_set_vec_1d[i * 3 + j] = off_set_vec[i][j];
            }
        }
        //int *d_off_set_vec_1d;
        hipMalloc((void**)&d_off_set_vec_1d, n_nebcells * 3 * sizeof(int));
        hipMemcpy(d_off_set_vec_1d, off_set_vec_1d, n_nebcells * 3 * sizeof(int), hipMemcpyHostToDevice);

    }

    /*
    * to build the neighbor list array, and initialize the parameters and allocate the memory of the device
    * variable: d_natoms: the number of atoms
    * variable: d_off_set_vec: the offset vector of each neighbor cell
    * variable: d_n_nebcells: the number of neighbor cells
    * variable: d_nebcell_list: the neighbor cell index of each cell
    * run kernel get_neb to get the neighbor cell index of each cell
    * run update(xyz) to build the neighbor list array
    */
    //void CellSearchGPU::search(std::vector<std::vector<float>> &xyz) {
    void CellSearchGPU::search() {
        auto start_time = std::chrono::high_resolution_clock::now();

        for (int i = 0; i < 3; ++i) {
            _cell_len[i] = _box_len[i]/_r_cutoff;
        }
        _ncells = _cell_len[0] * _cell_len[1] * _cell_len[2];

        hipMalloc((void**)&d_cell_len, 3 * sizeof(int));
        hipMemcpy(d_cell_len, _cell_len, 3 * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc((void**)&d_ncells, sizeof(int));
        hipMemcpy(d_ncells, &_ncells, sizeof(int), hipMemcpyHostToDevice);

        _natoms = _xyz.size();
        hipMalloc((void**)&d_natoms, sizeof(int));
        hipMemcpy(d_natoms, &_natoms, sizeof(int), hipMemcpyHostToDevice);

        
        
        hipMalloc((void**)&d_nebcell_list, _ncells * n_nebcells * sizeof(int));
        int threadsPerBlock = 256;
        int blocksPerGrid = (_ncells + threadsPerBlock - 1) / threadsPerBlock;
        //////////
        auto neb_start = std::chrono::high_resolution_clock::now();
        get_neb<<<blocksPerGrid, threadsPerBlock>>>(d_off_set_vec_1d, d_ncells, d_n_nebcells, d_cell_len, d_nebcell_list);
        auto neb_end = std::chrono::high_resolution_clock::now();
        auto neb_duration = std::chrono::duration_cast<std::chrono::microseconds>(neb_end - neb_start);
        //std::cout << "Time taken by get nebcell: " << neb_duration.count() << " microseconds" << std::endl;
        //////////
        //hipFree(d_off_set_vec_1d);

        // int *d_head, *d_lscl, *d_atom_cellindex, *d_cell_atoms_count;
        hipMalloc((void**)&d_head, _ncells * sizeof(int));                      // head[i] is the first atom in cell i
        hipMalloc((void**)&d_lscl, _natoms * sizeof(int));                      // lscl is the atom linked list, lscl[i] is the next atom in the cell
        hipMalloc((void**)&d_atom_cellindex, _natoms * sizeof(int));            // atom in which cell
        hipMalloc((void**)&d_cell_atoms_count, _ncells * sizeof(int));          // cell_atoms_count[i] is the number of atoms in cell i
        
        _neighborListArray = new int[_natoms * nnebs];
        update(_xyz);

        hipFree(d_nebcell_list);
        auto end_time = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
        time_cost = duration.count();
        //std::cout << "Time taken by cuda: " << time_cost << " milliseconds" << std::endl;
    }
/*
    bool CellSearchGPU::judge_update(float *d_xyz_1d){
        if (pre_xyz.size() == 0 || _skin == 0){
            return true;
        }
        // pre_xyz to 1d array and copy to device
        float *pre_xyz_1d = new float[_natoms * 3];
        for (int i = 0; i < _natoms; ++i) {
            for (int j = 0; j< 3; ++j) {
                pre_xyz_1d[i * 3 + j] = pre_xyz[i][j];
            }
        }
        float *d_pre_xyz_1d, *d_dis;
        hipMalloc((void**)&d_pre_xyz_1d, _natoms * 3 * sizeof(float));
        hipMemcpy(d_pre_xyz_1d, pre_xyz_1d, _natoms * 3 * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc((void**)&d_dis, _natoms * sizeof(float));
        // kernal calc_vec_distance
        int threadsPerBlock = 256;
        int blocksPerGrid = (_natoms + threadsPerBlock - 1) / threadsPerBlock;
        calc_vec_distance<<<blocksPerGrid, threadsPerBlock>>>(d_pre_xyz_1d, d_xyz_1d, d_dis, d_natoms);
        // copy d_dis to host
        float *dis = new float[_natoms];
        hipMemcpy(dis, d_dis, _natoms * sizeof(float), hipMemcpyDeviceToHost);
        
        delete[] pre_xyz_1d;
        hipFree(d_pre_xyz_1d);
        // judge whether to update
        float half_skin = _skin  * 0.5;
        for (int i = 0; i < _natoms; i++)
        {
            if (dis[i] > half_skin){
                delete[] dis;
                hipFree(d_dis);
                return true;
            }
        }
        delete[] dis;
        hipFree(d_dis);
        return false;
    }
*/
    /*
    * to update the neighbor list array, and allocate the memory of the device
    * variable: d_xyz_1d: the coordinates of all atoms
    * variable: d_head: the head atom of each cell
    * variable: d_lscl: the next atom of each cell
    * variable: d_atom_cellindex: the cell index of each atom
    * variable: d_cell_atoms_count: the number of atoms in each cell
    * run kernel build_linked_list_kernel to build the linked list
    * run kernel buildListArray to build the neighbor list array
    */
    void CellSearchGPU::update(const std::vector<std::vector<float>> &xyz) {
        float *xyz_1d = new float[_natoms * 3];
        for (int i = 0; i < _natoms; ++i) {
            for (int j = 0; j< 3; ++j) {
                xyz_1d[i * 3 + j] = xyz[i][j];
            }
        }
        float *d_xyz_1d;
        hipMalloc((void**)&d_xyz_1d, _natoms * 3 * sizeof(float));
        hipMemcpy(d_xyz_1d, xyz_1d, _natoms * 3 * sizeof(float), hipMemcpyHostToDevice);

        // judge whether to update the neighbor list array
        //bool is_update = judge_update(d_xyz_1d);
        bool is_update = true;
        if (!is_update) return;
        pre_xyz = xyz;

        float *d_wrap_xyz_1d;
        hipMalloc((void**)&d_wrap_xyz_1d, _natoms * 3 * sizeof(float));
        int threadsPerBlock = 256;
        int blocksPerGrid = (_natoms + threadsPerBlock - 1) / threadsPerBlock;
        // kernal wrap_coord
        wrap_coord<<<blocksPerGrid, threadsPerBlock>>>(d_xyz_1d, d_wrap_xyz_1d, d_box_len, d_natoms);

        // int *d_head, *d_lscl, *d_atom_cellindex, *d_cell_atoms_count;
        _cell_atoms_count = new int[_ncells];
        for (int i = 0; i < _ncells; ++i) {
            _cell_atoms_count[i] = 0;
        }
        hipMemcpy(d_cell_atoms_count, _cell_atoms_count, _ncells * sizeof(int), hipMemcpyHostToDevice);
        // auto start_time = std::chrono::high_resolution_clock::now();
        // int threadsPerBlock = 256;
        // int blocksPerGrid = (_natoms + threadsPerBlock - 1) / threadsPerBlock;

        //////////
        auto linked_list_start = std::chrono::high_resolution_clock::now();
        build_linked_list_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_wrap_xyz_1d, d_head, d_lscl, d_atom_cellindex, d_cell_atoms_count, d_r_cutoff, d_cell_len, d_natoms);
        auto linked_list_end = std::chrono::high_resolution_clock::now();
        auto linked_list_duration = std::chrono::duration_cast<std::chrono::microseconds>(linked_list_end - linked_list_start);
        //std::cout << "Time taken by build_linked_list: " << linked_list_duration.count() << " microseconds" << std::endl;
        //////////

        float r_cutoff2 = _r_cutoff * _r_cutoff;
        float *d_r_cutoff2;
        hipMalloc((void**)&d_r_cutoff2, sizeof(float));
        hipMemcpy(d_r_cutoff2, &r_cutoff2, sizeof(float), hipMemcpyHostToDevice);

        // int *temp_neighborListArray = new int[_natoms * nnebs]; // neighborListArray is the neighbor list of atoms, has a maximum 100 neighbors for each atom
        // for (int i = 0; i < _natoms * nnebs; ++i) {
        //     temp_neighborListArray[i] = -1;
        // }
        int arraySize = _natoms * nnebs;
        int *d_neighborListArray;
        hipMalloc((void**)&d_neighborListArray, arraySize * sizeof(int));
        
        blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;

        // Launch kernel to initialize the array
        initializeArray<<<blocksPerGrid, threadsPerBlock>>>(d_neighborListArray, arraySize);
        hipDeviceSynchronize(); // Ensure kernel execution is complete before proceeding


        // blocksPerGrid = (_natoms + threadsPerBlock - 1) / threadsPerBlock;

        //////////
        auto build_start = std::chrono::high_resolution_clock::now();
        buildListArray<<<blocksPerGrid, threadsPerBlock>>>(d_natoms, d_wrap_xyz_1d, d_head, d_lscl, d_atom_cellindex, d_cell_atoms_count, d_nebcell_list, d_cell_len, d_box_len, d_r_cutoff2, d_neighborListArray, nnebs);
        hipDeviceSynchronize();
        auto build_end = std::chrono::high_resolution_clock::now();
        auto build_duration = std::chrono::duration_cast<std::chrono::milliseconds>(build_end - build_start);
        //std::cout << "Time taken by build list array: " << build_duration.count() << " milliseconds" << std::endl;
        //////////

        //////////
        auto copy_start = std::chrono::high_resolution_clock::now();
        hipMemcpy(_neighborListArray, d_neighborListArray, _natoms * nnebs * sizeof(int), hipMemcpyDeviceToHost);
        auto copy_end = std::chrono::high_resolution_clock::now();
        auto copy_duration = std::chrono::duration_cast<std::chrono::milliseconds>(copy_end - copy_start);
        //std::cout << "Time taken by Memcopy to host: " << copy_duration.count() << " milliseconds" << std::endl;
        //////////

        hipFree(d_r_cutoff2);
        hipFree(d_xyz_1d);
        hipFree(d_wrap_xyz_1d);
        hipFree(d_neighborListArray);
        delete[] xyz_1d;
        // delete[] temp_neighborListArray;
        // out();
    }

    // return the neighbor list array
    std::vector<std::vector<int>> CellSearchGPU::get_particle_neighbor_info() {
        std::vector<std::vector<int>> neighborListArray;
        for (int i = 0; i < _natoms; ++i) {
            std::vector<int> neighborListArrayi;
            for(int j = 0; j < nnebs; ++j) {
                int neighborListArrayj = _neighborListArray[i * nnebs + j];
                if (neighborListArrayj >= 0 && neighborListArrayj != i) neighborListArrayi.push_back(neighborListArrayj);
            }
            neighborListArray.push_back(neighborListArrayi);
        }
        return neighborListArray;
    }

    CellSearchGPU::~CellSearchGPU()
    {
        hipFree(d_box_len);
        hipFree(d_cell_len);
        hipFree(d_ncells);
        hipFree(d_natoms);
        hipFree(d_nebcell_list);
        hipFree(d_head);
        hipFree(d_lscl);
        hipFree(d_atom_cellindex);
        hipFree(d_cell_atoms_count);
        delete[] _cell_atoms_count;
        delete[] _neighborListArray;
    }

}
